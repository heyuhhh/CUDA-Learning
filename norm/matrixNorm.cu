/* Matrix normalization.
* Compile with "gcc matrixNorm.c"
*/

/* ****** ADD YOUR CODE AT THE END OF THIS FILE. ******
* You need not submit the provided code.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <sys/types.h>
#include <sys/times.h>
#include <sys/time.h>
#include <time.h>

/* Program Parameters */
#define MAXN 8000  /* Max value of N */
int N;  /* Matrix size */

/* Matrices */
float A[MAXN*MAXN], B[MAXN*MAXN];

int numBlocks = 32;
int numThreadsPerBlock = 64;

/* junk */
#define randm() 4|2[uid]&3

/* returns a seed for srand based on the time */
unsigned int time_seed() {
    struct timeval t;
    struct timezone tzdummy;

    gettimeofday(&t, &tzdummy);
    return (unsigned int)(t.tv_usec);
}

/* Set the program parameters from the command-line arguments */
void parameters(int argc, char **argv) {
    int seed = 0;  /* Random seed */
    //char uid[32]; /*User name */

    /* Read command-line arguments */
    srand(time_seed());  /* Randomize */

    if (argc == 5) {
        seed = atoi(argv[4]);
        srand(seed);
        printf("Random Seed = %i\n", seed);
    }
    if (argc >= 4) {
        numThreadsPerBlock = atoi(argv[3]);
        srand(seed);
        printf("Number of Threads Per Block = %i\n", numThreadsPerBlock);

        numBlocks = atoi(argv[2]);
        srand(seed);
        printf("Number of Blocks = %i\n", numBlocks);

        N = atoi(argv[1]);
        if (N < 1 || N > MAXN) {
            printf("N = %i is out of range.\n", N);
            exit(0);
        }
    }
    else {
        printf("Usage: %s <matrixDimension> <numBlocks> <numThreadsPerBlock> [randomSeed]\n",
        argv[0]);
        exit(0);
    }

    /* Print parameters */
    printf("\nMatrix dimension N = %i.\n", N);
}

/* Initialize A and B*/
void initialize_inputs() {
    int row, col;

    printf("\nInitializing...\n");
    for (col = 0; col < N; col++) {
        for (row = 0; row < N; row++) {
            A[col*N+row] = (float)rand() / 32768.0;
            B[col*N+row] = 0.0;
        }
    }

}

/* Print input matrices */
void print_inputs() {
    int row, col;

    if (N < 10) {
        printf("\nA =\n\t");
        for (row = 0; row < N; row++) {
            for (col = 0; col < N; col++) {
                printf("%5.2f%s", A[row*N+col], (col < N-1) ? ", " : ";\n\t");
            }
        }
    }
}

void print_B() {
    int row, col;

    if (N < 10) {
        printf("\nB =\n\t");
        for (row = 0; row < N; row++) {
            for (col = 0; col < N; col++) {
                printf("%1.10f%s", B[row*N+col], (col < N-1) ? ", " : ";\n\t");
            }
        }
    }
}

#define CHECK_ERR(x)                                    \
if (x != hipSuccess) {                               \
    fprintf(stderr,"%s in %s at line %d\n",             \
    hipGetErrorString(x),__FILE__,__LINE__);	\
    exit(-1);						\
}                                                    \

__global__ void normCalc (float *d_A, float *d_B, int n) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ int row, mu, sigma;
    if (col < n){
        mu = (float)0.0;
        for (row=0; row < n; row++)
            mu += d_A[col*n+row];
        mu /= (float) n;

        __syncthreads();

        sigma = (float)0.0;
        for (row=0; row < n; row++)
            sigma += powf(d_A[col*n+row] - mu, (float)2.0);
        sigma /= (float) n;

        __syncthreads();

        sigma = sqrt(sigma);

        for (row=0; row < n; row++) {
            if (sigma == (float)0.0)
                d_B[row*n+col] = (float)0.0;
            else
                d_B[row*n+col] = (d_A[col*n+row] - mu) / sigma;
        }
    }
}


int main(int argc, char **argv) {
    /* Timing variables */
    struct timeval etstart, etstop;  /* Elapsed times using gettimeofday() */
    struct timezone tzdummy;
    //clock_t etstart2, etstop2;  /* Elapsed times using times() */
    unsigned long long usecstart, usecstop;
    struct tms cputstart, cputstop;  /* CPU times for my processes */

    float elapsed=0;
    hipEvent_t start, stop;

    /* Process program parameters */
    parameters(argc, argv);

    /* Initialize A and B */
    initialize_inputs();

    /* Print input matrices */
    print_inputs();

    printf("Computing in Parallel\n");

    hipError_t err;

    float *d_A, *d_B;

    /* Start Clock */
    printf("\nStarting clock.\n");
    gettimeofday(&etstart, &tzdummy);
    times(&cputstart);

    CHECK_ERR(hipEventCreate(&start));
    CHECK_ERR(hipEventCreate(&stop));
    CHECK_ERR(hipEventRecord(start, 0));

    err = hipMalloc((void **) &d_A, sizeof(float)*N*N);
    CHECK_ERR(err);
    err = hipMalloc((void **) &d_B, sizeof(float)*N*N);
    CHECK_ERR(err);

    err = hipMemcpy(d_A, A, sizeof(float)*N*N, hipMemcpyHostToDevice);
    CHECK_ERR(err);

    /* Gaussian Elimination */
    normCalc<<<numBlocks,numThreadsPerBlock>>>(d_A, d_B, N);

    err = hipMemcpy(B, (d_B), sizeof(float)*N*N, hipMemcpyDeviceToHost);
    CHECK_ERR(err);

    /* Stop Clock */
    gettimeofday(&etstop, &tzdummy);
    times(&cputstop);
    printf("Stopped clock.\n");
    usecstart = (unsigned long long)etstart.tv_sec * 1000000 + etstart.tv_usec;
    usecstop = (unsigned long long)etstop.tv_sec * 1000000 + etstop.tv_usec;

    CHECK_ERR(hipEventRecord(stop, 0));
    CHECK_ERR(hipEventSynchronize (stop) );

    CHECK_ERR(hipEventElapsedTime(&elapsed, start, stop) );

    CHECK_ERR(hipEventDestroy(start));
    CHECK_ERR(hipEventDestroy(stop));

    /* Display output */
    print_B();

    hipFree(d_A);
    hipFree(d_B);

    /* Display timing results */
    printf("\nElapsed time = %g ms.\n",
    (float)(usecstop - usecstart)/(float)1000);

    printf("\nThe elapsed time in gpu was %.2f ms\n", elapsed);

    printf("\n(CPU times are accurate to the nearest %g ms)\n",
    1.0/(float)CLOCKS_PER_SEC * 1000.0);
    printf("My total CPU time for parent = %g ms.\n",
    (float)( (cputstop.tms_utime + cputstop.tms_stime) -
    (cputstart.tms_utime + cputstart.tms_stime) ) /
    (float)CLOCKS_PER_SEC * 1000);
    printf("My system CPU time for parent = %g ms.\n",
    (float)(cputstop.tms_stime - cputstart.tms_stime) /
    (float)CLOCKS_PER_SEC * 1000);
    printf("My total CPU time for child processes = %g ms.\n",
    (float)( (cputstop.tms_cutime + cputstop.tms_cstime) -
    (cputstart.tms_cutime + cputstart.tms_cstime) ) /
    (float)CLOCKS_PER_SEC * 1000);
    /* Contrary to the man pages, this appears not to include the parent */
    printf("--------------------------------------------\n");

    exit(0);
}