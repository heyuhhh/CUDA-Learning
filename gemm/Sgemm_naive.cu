#include "hip/hip_runtime.h"
#include <cstdio>
#include <stdlib.h>
#include <float.h>
#include <hip/hip_runtime.h>

#define OFFSET(row, col, ld) ((row) * (ld) + (col))
#define FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

float testError(void);

float testPerformance(
    void (*gpuGemm) (float*, float*, float*, const int, const int, const int),
    dim3 gridDim, dim3 blockDim, const int M, const int N, const int K, const int repeat
);

void cpuSgemm(
    float *a, float *b, float *c, const int M, const int N, const int K) {

    for (int m = 0; m < M; m++) {
        for (int n = 0; n < N; n++) {
            float psum = 0.0;
            for (int k = 0; k < K; k++) {
                psum += a[OFFSET(m, k, K)] * b[OFFSET(k, n, N)];
            }
            c[OFFSET(m, n, N)] = psum;
        }
    }
}

__global__
void naiveSgemm(
    float* __restrict__ a, float* __restrict__ b, float* __restrict__ c,
    const int M, const int N, const int K) {
    
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    int m = blockIdx.y * blockDim.y + threadIdx.y;

    if (m < M && n < N) {
        float psum = 0;
        for (int k = 0; k < K; k++) {
            psum += a[OFFSET(m, k, K)] * b[OFFSET(k, n, N)];
        }
        c[OFFSET(m, n, N)] = psum;
    }
}

int main() {
    float max_error = testError();
    printf("Max Error = %f\n", max_error);

    printf("\nKernal = naiveSgemm\n");
    const int M_list[15] = {128, 192, 256, 384, 512, 768, 1024, 1536, 2048, 3072, 4096, 6144, 8192, 12288, 16384};
    const int N_list[15] = {128, 192, 256, 384, 512, 768, 1024, 1536, 2048, 3072, 4096, 6144, 8192, 12288, 16384};
    const int K_list[15] = {1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024, 1024};
    
    const int outer_repeat = 10, inner_repeat = 1;
    const int BM = 32, BN = 32;
    void (*gpuGemm) (float *, float *, float *, const int, const int, const int) = naiveSgemm;
    const int TESTNUM = 15;

    for (int i = 0; i < TESTNUM; i++) {
        const int M = M_list[i], N = N_list[i], K = K_list[i];

        dim3 blockDim(BN, BM);
        dim3 gridDim((N + BN - 1) / BN, (M + BM - 1) / BM);

        double max_sec = 0.0;
        double min_sec = DBL_MAX;
        double total_sec = 0.0;

        for (int j = 0; j < outer_repeat; j++) {
            double this_sec = testPerformance(gpuGemm, gridDim, blockDim, M, N, K, inner_repeat);
            max_sec = max(max_sec, this_sec);
            min_sec = min(min_sec, this_sec);
            total_sec += this_sec;
        }

        double avg_sec = total_sec / outer_repeat;
        // 有效带宽：计算量 / 时间
        // (/ 1024 / 1024 / 1024) FLOPS->GFLOPS
        double avg_Gflops = ((double)M) * N * K * 2 / 1024 / 1024 / 1024 / avg_sec;

        printf("M N K = %6d %6d %6d, Time = %12.8lf %12.8lf %12.8lf s, AVG Performance = %10.4lf Gflops\n", M, N, K, min_sec, avg_sec, max_sec, avg_Gflops);
    }
    return 0;
}

float testError(void) {
    const int BM = 32, BN = 32;
    const int M = 512, N = 512, K = 512;
    dim3 blockDim(BN, BM);
    dim3 gridDim((N + BN - 1) / BN, (M + BM - 1) / BM);

    size_t size_a = M * K * sizeof(float);
    size_t size_b = K * N * sizeof(float);
    size_t size_c = M * N * sizeof(float);

    float *h_a, *h_b, *h_c, *d_a, *d_b, *d_c, *h_d_c;
    h_a = (float *)malloc(size_a);
    h_b = (float *)malloc(size_b);
    h_c = (float *)malloc(size_c);
    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);
    h_d_c = (float *)malloc(size_c);

    srand(time(0));
    for (int i = 0; i < M * K; i++)
        h_a[i] = rand() / float(RAND_MAX);
    for (int i = 0; i < K * N; i++)
        h_b[i] = rand() / float(RAND_MAX);
    hipMemset(d_c, 15, size_c);

    cpuSgemm(h_a, h_b, h_c, M, N, K);

    hipMemcpy(d_a, h_a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size_b, hipMemcpyHostToDevice);
    naiveSgemm<<<gridDim, blockDim>>>(d_a, d_b, d_c, M, N, K);
    hipMemcpy(h_d_c, d_c, size_c, hipMemcpyDeviceToHost);

    float max_error = 0.0;
    for (int i = 0; i < M * N; i++) {
        float this_error = abs(h_d_c[i] - h_c[i]);
        if (max_error != max_error || this_error != this_error) // nan
            max_error = -NAN;
        else
            max_error = max(max_error, this_error);
    }

    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_d_c);

    return max_error;
}

float testPerformance(
    void (*gpuGemm) (float *, float *, float *, const int, const int, const int),
    dim3 gridDim, dim3 blockDim, const int M, const int N, const int K, const int repeat) {
    size_t size_a = M * K * sizeof(float);
    size_t size_b = K * N * sizeof(float);
    size_t size_c = M * N * sizeof(float);

    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start);
    for (int i = 0; i < repeat; i++) {
        gpuGemm<<<gridDim, blockDim>>>(d_a, d_b, d_c, M, N, K);
    }
    hipEventRecord(end);
    hipEventSynchronize(end);

    float msec, sec;
    hipEventElapsedTime(&msec, start, end);
    sec = msec / 1000 / repeat;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return sec;
}