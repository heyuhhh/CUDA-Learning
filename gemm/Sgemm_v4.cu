#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <float.h> // float4
#include <hip/hip_fp16.h> // half
#include <mma.h> // wmma
using namespace nvcuda;

#define OFFSET(row, col, ld) ((row) * (ld) + (col))
#define FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

float testError(void (*gpuGemm) (half*, half*, half*, const int, const int, const int), const int, const int, const int);

float testPerformance(
    void (*gpuGemm) (half*, half*, half*, const int, const int, const int),
    dim3 gridDim, dim3 blockDim, const int M, const int N, const int K, const int repeat
);

void cpuSgemm(
    half *a, half *b, half *c, const int M, const int N, const int K) {

    for (int m = 0; m < M; m++) {
        for (int n = 0; n < N; n++) {
            float psum = 0.0;
            for (int k = 0; k < K; k++) {
                psum += (float)a[OFFSET(m, k, K)] * (float)b[OFFSET(k, n, N)];
            }
            c[OFFSET(m, n, N)] = (half)psum;
        }
    }
}

__global__ void myHGEMMAlignedV1(half* __restrict__ a, half* __restrict__ b, 
                                half* __restrict__ c, const int M, const int N, const int K) {
    const int BM = 128;
    const int BN = 256;
    const int BK = 32;

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tid = threadIdx.x;
    
    const int APAD = 8;
    const int BPAD = 8;

    __shared__ half s_a[BM][BK + APAD];
    __shared__ half s_b[BK][BN + BPAD];

    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> frag_a[2][4];
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> frag_b[2][4];
    wmma::fragment<wmma::accumulator, 16, 16, 16, half> frag_c[4][4];

    for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 4; j++) {
            wmma::fill_fragment(frag_c[i][j], 0.0);
        }
    }

    int load_a_smem_m = (tid >> 2) << 1; // 每个线程读取相邻两行的16个元素，每一行8个
    int load_a_smem_k = (tid & 3) << 3;
    int load_b_smem_k = (tid >> 5) << 2; // 还是读 8 个，现在要读4行
    int load_b_smem_n = (tid & 31) << 3;

    int load_a_gmem_m = by * BM + load_a_smem_m;
    int load_b_gmem_n = bx * BN + load_b_smem_n;

    int load_a_gmem_addr = OFFSET(load_a_gmem_m, load_a_smem_k, K); // 现在是每个线程块会遍历矩阵的一行进行计算
    int load_b_gmem_addr = OFFSET(load_b_smem_k, load_b_gmem_n, N);

    /*
    求出每个线程束所负责的 行、列
    现在一个block大小为 128*256，一共 256 个线程，8个线程束，每个线程负责 64*32*64 的矩阵乘法
    将 128*256 的矩阵能划分为八个区域，上下各四个区域
        ----------------
        0 \ 1 \ 2 \ 3
        ----------------
        5 \ 6 \ 7 \ 8 
        ----------------
    */    

    int comp_c_frag_m = (tid >> 5) &  1;
    int comp_c_frag_n = (tid >> 5) >> 1;

    for (int bk = 0; bk < K / BK; bk++) {
        // 注意数据都是 half 类型（即 fp16），float4 会读取 128 bit，即一次就能读 8 个数
        FLOAT4(s_a[load_a_smem_m    ][load_a_smem_k]) = FLOAT4(a[load_a_gmem_addr        ]);
        FLOAT4(s_a[load_a_smem_m + 1][load_a_smem_k]) = FLOAT4(a[load_a_gmem_addr +     K]);
        FLOAT4(s_b[load_b_smem_k    ][load_b_smem_n]) = FLOAT4(b[load_b_gmem_addr        ]);
        FLOAT4(s_b[load_b_smem_k + 1][load_b_smem_n]) = FLOAT4(b[load_b_gmem_addr +     N]);
        FLOAT4(s_b[load_b_smem_k + 2][load_b_smem_n]) = FLOAT4(b[load_b_gmem_addr + 2 * N]);
        FLOAT4(s_b[load_b_smem_k + 3][load_b_smem_n]) = FLOAT4(b[load_b_gmem_addr + 3 * N]);

        load_a_gmem_addr += BK;
        load_b_gmem_addr += BK * N;

        __syncthreads();

        // 接下来使用 load_matrix_sync 以线程束为单位读取对应的矩阵，每次读一个 16*16 的矩阵
        // 每个线程束会读取 64*32 的矩阵，对于 frag 来说就是 2*4 大小的
        wmma::load_matrix_sync(frag_a[0][0], &s_a[comp_c_frag_m * 64     ][ 0], BK + APAD);
        wmma::load_matrix_sync(frag_a[0][1], &s_a[comp_c_frag_m * 64 + 16][ 0], BK + APAD);
        wmma::load_matrix_sync(frag_a[0][2], &s_a[comp_c_frag_m * 64 + 32][ 0], BK + APAD);
        wmma::load_matrix_sync(frag_a[0][3], &s_a[comp_c_frag_m * 64 + 48][ 0], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][0], &s_a[comp_c_frag_m * 64     ][16], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][1], &s_a[comp_c_frag_m * 64 + 16][16], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][2], &s_a[comp_c_frag_m * 64 + 32][16], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][3], &s_a[comp_c_frag_m * 64 + 48][16], BK + APAD);

        wmma::load_matrix_sync(frag_b[0][0], &s_b[ 0][comp_c_frag_n * 64     ], BN + BPAD);
        wmma::load_matrix_sync(frag_b[0][1], &s_b[ 0][comp_c_frag_n * 64 + 16], BN + BPAD);
        wmma::load_matrix_sync(frag_b[0][2], &s_b[ 0][comp_c_frag_n * 64 + 32], BN + BPAD);
        wmma::load_matrix_sync(frag_b[0][3], &s_b[ 0][comp_c_frag_n * 64 + 48], BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][0], &s_b[16][comp_c_frag_n * 64     ], BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][1], &s_b[16][comp_c_frag_n * 64 + 16], BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][2], &s_b[16][comp_c_frag_n * 64 + 32], BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][3], &s_b[16][comp_c_frag_n * 64 + 48], BN + BPAD);

        for (int i = 0; i < 4; i++) {
            for (int j = 0; j < 4; j++) {
                wmma::mma_sync(frag_c[i][j], frag_a[0][i], frag_b[0][j], frag_c[i][j]);
                wmma::mma_sync(frag_c[i][j], frag_a[1][i], frag_b[1][j], frag_c[i][j]);
            }
        }
        __syncthreads();
    }

    int store_c_gmem_m = by * BM + comp_c_frag_m * 64;
    int store_c_gmem_n = bx * BN + comp_c_frag_n * 64;
    int store_c_gmem_addr = OFFSET(store_c_gmem_m, store_c_gmem_n, N);

    for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 4; j++) {
            wmma::store_matrix_sync(&c[store_c_gmem_addr + i * 16 * N + j * 16], frag_c[i][j], N, wmma::mem_row_major);        
        }
    }
}

int main() {
    void (*gpuGemm) (half *, half *, half *, const int, const int, const int) = myHGEMMAlignedV1;
    float max_error = testError(gpuGemm, 256, 256, 256);
    printf("Max Error = %f\n", max_error);

    printf("\nKernal = Sgemm_v4\n");
    const int TESTNUM = 7;
    const int M_list[TESTNUM] = {256, 512, 1024, 2048, 4096, 8192, 16384};
    const int N_list[TESTNUM] = {256, 512, 1024, 2048, 4096, 8192, 16384};
    const int K_list[TESTNUM] = {256, 512, 1024, 2048, 4096, 8192, 16384};
    
    const int outer_repeat = 10, inner_repeat = 1;
    

    const int BM = 128, BN = 256;
    for (int i = 0; i < TESTNUM; i++) {
        const int M = M_list[i], N = N_list[i], K = K_list[i];
        dim3 blockDim(256);
        dim3 gridDim((N + BN - 1) / BN, (M + BM - 1) / BM);

        double max_sec = 0.0;
        double min_sec = DBL_MAX;
        double total_sec = 0.0;

        for (int j = 0; j < outer_repeat; j++) {
            double this_sec = testPerformance(gpuGemm, gridDim, blockDim, M, N, K, inner_repeat);
            max_sec = max(max_sec, this_sec);
            min_sec = min(min_sec, this_sec);
            total_sec += this_sec;
        }

        double avg_sec = total_sec / outer_repeat;
        // 有效带宽：计算量 / 时间
        // (/ 1024 / 1024 / 1024) FLOPS->GFLOPS
        double avg_Gflops = ((double)M) * N * K * 2 / 1024 / 1024 / 1024 / avg_sec;

        printf("M N K = %6d %6d %6d, Time = %12.8lf %12.8lf %12.8lf s, AVG Performance = %10.4lf Gflops\n", M, N, K, min_sec, avg_sec, max_sec, avg_Gflops);
    }
    return 0;
}

float testError(
    void (*gpuF16F16Gemm) (half *, half *, half *, int, int, int),
    const int M, const int N, const int K) {

    size_t size_a = M * K * sizeof(half);
    size_t size_b = K * N * sizeof(half);
    size_t size_c = M * N * sizeof(half);

    half *h_a, *h_b, *d_a, *d_b;
    half *h_c, *d_c, *h_d_c;
    h_a = (half *)malloc(size_a);
    h_b = (half *)malloc(size_b);
    h_c = (half *)malloc(size_c);
    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);
    h_d_c = (half *)malloc(size_c);

    srand(time(0));
    for (int i = 0; i < M * K; i++)
        h_a[i] = (half)(rand() / float(RAND_MAX));
    for (int i = 0; i < K * N; i++)
        h_b[i] = (half)(rand() / float(RAND_MAX));

    cpuSgemm(h_a, h_b, h_c, M, N, K);

    const int BM = 128, BN = 256;
    dim3 blockDim(256);
    dim3 gridDim((N + BN - 1) / BN, (M + BM - 1) / BM);

    hipMemcpy(d_a, h_a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size_b, hipMemcpyHostToDevice);
    gpuF16F16Gemm<<<gridDim, blockDim>>>(d_a, d_b, d_c, M, N, K);
    printf("%s\n", hipGetErrorString(hipGetLastError()));
    hipMemcpy(h_d_c, d_c, size_c, hipMemcpyDeviceToHost);

    float max_error = 0.0;
    for (int i = 0; i < M * N; i++) {
        float this_error = abs((float)h_d_c[i] - (float)h_c[i]);
        if (max_error != max_error || this_error != this_error) // nan
            max_error = -NAN;
        else
            max_error = max(max_error, this_error);
    }

    free(h_a); free(h_b); free(h_c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c); free(h_d_c);

    return max_error;
}

float testPerformance(
    void (*gpuGemm) (half *, half *, half *, const int, const int, const int),
    dim3 gridDim, dim3 blockDim, const int M, const int N, const int K, const int repeat) {
    size_t size_a = M * K * sizeof(half);
    size_t size_b = K * N * sizeof(half);
    size_t size_c = M * N * sizeof(half);

    half *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start);
    for (int i = 0; i < repeat; i++) {
        gpuGemm<<<gridDim, blockDim>>>(d_a, d_b, d_c, M, N, K);
    }
    hipEventRecord(end);
    hipEventSynchronize(end);

    float msec, sec;
    hipEventElapsedTime(&msec, start, end);
    sec = msec / 1000 / repeat;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return sec;
}